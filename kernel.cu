/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
    

__global__ void VecAdd(int n, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (1 * n) vector
     *   where B is a (1 * n) vector
     *   where C is a (1 * n) vector
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    int index=blockIdx.x*blockDim.x+threadIdx.x;
    if(index<n){
    C[index]=A[index]+B[index];}
     
}


void basicVecAdd( float *A,  float *B, float *C, int n)
{

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 512; 
    //INSERT CODE HERE
	int blocksize = 256;
    VecAdd<<<(n-1)/blocksize+1,blocksize>>>(n,A,B,C);
}

