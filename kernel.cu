/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
    

__global__ void VecAdd(int n, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (1 * n) vector
     *   where B is a (1 * n) vector
     *   where C is a (1 * n) vector
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    int index=blockIdx.x*blockDim.x+threadIdx.x;
    if(index<n){
    C[index]=A[index]+B[index];}
     
}


/*void basicVecAdd(float *A_h,float *B_h, float *C_h, unsigned VecSize)
 {  // printf("lunch kernel");
    float *A0_d, *B0_d, *C0_d;
    float *A1_d, *B1_d, *C1_d;
    float *A2_d, *B2_d, *C2_d;
    float *A3_d, *B3_d, *C3_d;
    float *A_d, *B_d, *C_d;
    int SegSize=VecSize/4;
    cudaMalloc((void**)&A0_d,sizeof(float)*SegSize);
    cudaMalloc((void**)&B0_d,sizeof(float)*SegSize);
    cudaMalloc((void**)&C0_d,sizeof(float)*SegSize);

    cudaMalloc((void**)&A1_d,sizeof(float)*SegSize);
    cudaMalloc((void**)&B1_d,sizeof(float)*SegSize);
    cudaMalloc((void**)&C1_d,sizeof(float)*SegSize);

    cudaMalloc((void**)&A2_d,sizeof(float)*SegSize);
    cudaMalloc((void**)&B2_d,sizeof(float)*SegSize);
    cudaMalloc((void**)&C2_d,sizeof(float)*SegSize);

    cudaMalloc((void**)&A3_d,sizeof(float)*SegSize);
    cudaMalloc((void**)&B3_d,sizeof(float)*SegSize);
    cudaMalloc((void**)&C3_d,sizeof(float)*SegSize);


    cudaMalloc((void**)&A_d,sizeof(float)*4*SegSize);
    cudaMalloc((void**)&B_d,sizeof(float)*4*SegSize);
    cudaMalloc((void**)&C_d,sizeof(float)*4*SegSize);

    cudaDeviceSynchronize();
    // Initialize thread block and kernel grid dimensions ---------------------
    cudaStream_t stream0,stream1,stream2,stream3;
    cudaStreamCreate(&stream0);
    cudaStreamCreate(&stream1);
    cudaStreamCreate(&stream2);
    cudaStreamCreate(&stream3);
   
    // Timer timer;
    // startTime(&timer);

     int blocksize=256;
     int num=VecSize/(SegSize*4);
     int i=0;
    //INSERT CODE HERE
    for(int j=0;j<num;j++){

             cudaMemcpyAsync(A0_d,A_h+i,SegSize*sizeof(float),cudaMemcpyHostToDevice,stream0);
             cudaMemcpyAsync(B0_d,B_h+i,SegSize*sizeof(float),cudaMemcpyHostToDevice,stream0);
             cudaMemcpyAsync(A1_d,A_h+i+SegSize,SegSize*sizeof(float),cudaMemcpyHostToDevice,stream1);
             cudaMemcpyAsync(B1_d,B_h+i+SegSize,SegSize*sizeof(float),cudaMemcpyHostToDevice,stream1);
            
            VecAdd<<<(SegSize-1)/blocksize+1,blocksize,0,stream0>>>(SegSize,A0_d,B0_d,C0_d);
            VecAdd<<<(SegSize-1)/blocksize+1,blocksize,0,stream1>>>(SegSize,A1_d,B1_d,C1_d);
     
            cudaMemcpyAsync(C_h+i,C0_d,SegSize*sizeof(float),cudaMemcpyDeviceToHost,stream0);
            cudaMemcpyAsync(C_h+i+SegSize,C1_d,SegSize*sizeof(float),cudaMemcpyDeviceToHost,stream1);


             cudaMemcpyAsync(A2_d,A_h+i+2*SegSize,SegSize*sizeof(float),cudaMemcpyHostToDevice,stream2);
             cudaMemcpyAsync(B2_d,B_h+i+2*SegSize,SegSize*sizeof(float),cudaMemcpyHostToDevice,stream2);
             cudaMemcpyAsync(A3_d,A_h+i+3*SegSize,SegSize*sizeof(float),cudaMemcpyHostToDevice,stream3);
             cudaMemcpyAsync(B3_d,B_h+i+3*SegSize,SegSize*sizeof(float),cudaMemcpyHostToDevice,stream3);
        
 
            VecAdd<<<(SegSize-1)/blocksize+1,blocksize,0,stream2>>>(SegSize,A2_d,B2_d,C2_d);
            VecAdd<<<(SegSize-1)/blocksize+1,blocksize,0,stream3>>>(SegSize,A3_d,B3_d,C3_d);


            cudaMemcpyAsync(C_h+i+2*SegSize,C2_d,SegSize*sizeof(float),cudaMemcpyDeviceToHost,stream2);
            cudaMemcpyAsync(C_h+i+3*SegSize,C3_d,SegSize*sizeof(float),cudaMemcpyDeviceToHost,stream3);
            i+=4*SegSize;
      }
       if(i<VecSize){
           cudaMemcpy(A_d,A_h+i,sizeof(float)*(VecSize-i),cudaMemcpyHostToDevice);
           cudaMemcpy(B_d,B_h+i,sizeof(float)*(VecSize-i),cudaMemcpyHostToDevice);

             VecAdd<<<(VecSize-i-1)/blocksize+1,blocksize>>>(VecSize-i,A_d,B_d,C_d);


             cudaMemcpy(C_h+i,C_d,(VecSize-i)*sizeof(float),cudaMemcpyDeviceToHost);}
    
    //  stopTime(&timer);printf("\n Execution time is %f s\n",elapsedTime(timer));
   cudaFree(A0_d);
    cudaFree(B0_d);
    cudaFree(C0_d);
    cudaFree(A1_d);
    cudaFree(B1_d);
    cudaFree(C1_d);
   cudaFree(A2_d);
    cudaFree(B2_d);
    cudaFree(C2_d);
    cudaFree(A3_d);
    cudaFree(B3_d);
    cudaFree(C3_d);

    cudaFree(A_d);
    cudaFree(B_d);
    cudaFree(C_d);
    //INSERT CODE HERE
    
}*/

