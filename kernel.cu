
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE
__global__ void histogram_kernel(unsigned int* input, unsigned int* bin, unsigned int num_elements, unsigned int num_bins){
         extern __shared__ unsigned int histogram_private[];
          int j=threadIdx.x;
         while (j<num_bins) {
            histogram_private[j]=0;
            j+=blockDim.x;
           }
          __syncthreads();

          int i=threadIdx.x+blockIdx.x*blockDim.x;
          int stride=blockDim.x*gridDim.x;
          while(i<num_elements){

               atomicAdd(&(histogram_private[input[i]]),1);

               i+=stride;
          }
          __syncthreads();
          int k=threadIdx.x;
          while(k<num_bins){
               atomicAdd(&(bin[k]), histogram_private[k]);
               k+=blockDim.x;
          }

} 





















/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void histogram(unsigned int* input, unsigned int* bins, unsigned int num_elements,
        unsigned int num_bins) {

    // INSERT CODE HERE

          const unsigned int BLOCK_SIZE=512;
           
          histogram_kernel<<<BLOCK_SIZE,BLOCK_SIZE,num_bins*sizeof(unsigned int)>>>(input,bins,num_elements,num_bins);

















}


