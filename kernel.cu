
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 1024 

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE
__global__ void Scan1(float *out, float *in,unsigned in_size)
{
   __shared__ float XY[2*BLOCK_SIZE];
   int i=threadIdx.x;
   int j=(blockIdx.x*blockDim.x)*2;
   if(i+j<in_size&&i+j>0) {
        XY[threadIdx.x]=in[i+j-1];
   }else{
        XY[threadIdx.x]=0.0;     
   }
   if (i+j+blockDim.x<in_size){
        XY[threadIdx.x+blockDim.x]=in[i+j+blockDim.x-1];
   }else{
        XY[threadIdx.x+blockDim.x]=0.0;
   }
   __syncthreads();
   for(unsigned int stride=1;stride<=BLOCK_SIZE;stride*=2){
       int index=(threadIdx.x+1)*stride*2-1;
       if(index<2*BLOCK_SIZE){ XY[index]+=XY[index-stride];}
       __syncthreads();


    }

    for(unsigned int stride=BLOCK_SIZE/2;stride>0;stride/=2){
        __syncthreads();
        int index=(threadIdx.x+1)*stride*2-1;
        if(index+stride<2*BLOCK_SIZE){
           XY[index+stride]+=XY[index];
         }

     }
     __syncthreads();
     if(i+j<in_size) out[i+j]=XY[threadIdx.x];
     if(i+j+blockDim.x<in_size) out[i+j+blockDim.x]=XY[i+blockDim.x];
     __syncthreads();
     if(i==BLOCK_SIZE-1&&j+i<in_size) in[blockIdx.x]=XY[i+blockDim.x];
     __syncthreads();

}

__global__ void Scan2(float *out, float *in,unsigned in_size)
{
   __shared__ float XY[2*BLOCK_SIZE];
   int i=threadIdx.x;
   int j=(blockIdx.x*blockDim.x)*2;
   if(i+j<in_size) {
        XY[threadIdx.x]=in[i+j];
    }else{
        XY[threadIdx.x]=0.0;
   }
   if (i+j+blockDim.x<in_size){
        XY[threadIdx.x+blockDim.x]=in[i+j+blockDim.x];
   }else{
        XY[threadIdx.x+blockDim.x]=0.0;
   }


   __syncthreads();
   for(unsigned int stride=1;stride<=BLOCK_SIZE;stride*=2){
       int index=(threadIdx.x+1)*stride*2-1;
       if(index<2*BLOCK_SIZE) XY[index]+=XY[index-stride];
       __syncthreads();


    }

    for(unsigned int stride=BLOCK_SIZE/2;stride>0;stride/=2){
        __syncthreads();
        int index=(threadIdx.x+1)*stride*2-1;
        if(index+stride<2*BLOCK_SIZE){
           XY[index+stride]+=XY[index];
         }

     }
     __syncthreads();
     if(i+j<in_size) out[i+j]=XY[threadIdx.x];
     if(i+j+blockDim.x<in_size) out[i+j+blockDim.x]=XY[i+blockDim.x];
    // if(i=BLOCK_SIZE-1) array[blockIdx.x]=out[j+i];
     __syncthreads();







}

__global__ void Add(float *arr_out, float *out, int in_size){

    __shared__ float XY[2*BLOCK_SIZE];
    __shared__ float ARR;

   int i=threadIdx.x;
   int j=(blockIdx.x*blockDim.x)*2;
   if(i+j<in_size) {
        XY[threadIdx.x]=out[i+j];
   }else{
        XY[threadIdx.x]=0.0;
   }
   if (i+j+blockDim.x<in_size){
        XY[threadIdx.x+blockDim.x]=out[i+j+blockDim.x];
   }else{
        XY[threadIdx.x+blockDim.x]=0.0;
   }
 //  __syncthreads();

   if(blockIdx.x>0) ARR=arr_out[blockIdx.x-1];
   __syncthreads();
    if(blockIdx.x>0){
      if(i+j<in_size)
          XY[threadIdx.x] +=ARR;
      if(i+j+blockDim.x<in_size)
          XY[threadIdx.x+blockDim.x]+=ARR;
      __syncthreads();
    }

   // if(blockIdx.x==1) XY[i]+=300;
    __syncthreads();
    if(i+j<in_size) out[i+j]=XY[threadIdx.x];
    if(i+j+blockDim.x<in_size) out[i+j+blockDim.x]=XY[threadIdx.x+blockDim.x];
    __syncthreads();



}









/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void preScan(float *out, float *in, unsigned in_size)
{
    // INSERT CODE HERE
  //  float *array_h,*arr_out_h;
   // float *array_d,*arr_out_d;
    int arr_size=(in_size/2-1)/BLOCK_SIZE+1;
   // array = (float*)calloc(arr_size, sizeof(float));
   // arr_out_h=(float*)calloc(arr_size,sizeof(float));
  //  cudaMalloc((void**)&array_,sizeof(float)*VecSize);
   // int arr_size=(in_size/2-1)/BLOCK_SIZE;
    Scan1<<<(in_size/2)/BLOCK_SIZE+1,BLOCK_SIZE>>>(out,in,in_size);
     // Scan<<<512,512>>>(out,in,in_size);
    Scan2<<<(arr_size/2)/BLOCK_SIZE+1,BLOCK_SIZE>>>(in,in,arr_size);

    Add<<<(in_size/2)/BLOCK_SIZE+1,BLOCK_SIZE>>>(in,out,in_size);


















}

