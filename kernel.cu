
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

__global__ void reduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    __shared__ float Sum[2*BLOCK_SIZE]; 
    unsigned int t=threadIdx.x;
    unsigned int start=2*blockIdx.x*blockDim.x;
    if(start+t<size){
      Sum[t]=in[start+t];
    }else{
      Sum[t]=0.0;
    }
    if(start+blockDim.x+t<size){
       Sum[blockDim.x+t]=in[start+blockDim.x+t];
     }else{
      Sum[blockDim.x+t]=0.0;
    }
    __syncthreads();
    for(unsigned int stride=BLOCK_SIZE;stride>=1;stride=stride/2){
           __syncthreads();
           if(threadIdx.x<stride){
              Sum[t]=Sum[t]+Sum[t+stride];
             }
     }
     __syncthreads();
     if(threadIdx.x==0){
        out[blockIdx.x]=Sum[0];}
    






























}
