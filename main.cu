/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.cu"

int main (int argc, char *argv[])
{
    //set standard seed
    srand(217);

    Timer timer;
   hipError_t cuda_ret;
   Timer timeExe;

//   float elapstimeforExe = 0;
    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;
   // float *A0_d, *B0_d, *C0_d;
   // float *A1_d, *B1_d, *C1_d;
    size_t A_sz, B_sz, C_sz;
    unsigned VecSize;
   
    dim3 dim_grid, dim_block;

    if (argc == 1) {
        VecSize = 1000000;

      } else if (argc == 2) {
      VecSize = atoi(argv[1]);   
      
      
      }
  
      else {
        printf("\nOh no!\nUsage: ./vecAdd <Size>");
        exit(0);
    }

    A_sz = VecSize;
    B_sz = VecSize;
    C_sz = VecSize;
 /*   A_h = (float*) malloc( sizeof(float)*A_sz );
    for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }

    B_h = (float*) malloc( sizeof(float)*B_sz );
    for (unsigned int i=0; i < B_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    C_h = (float*) malloc( sizeof(float)*C_sz );*/

    hipHostAlloc((void **) &A_h, A_sz*sizeof(float),hipHostMallocDefault);
    for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }

    hipHostAlloc((void **) &B_h, B_sz*sizeof(float),hipHostMallocDefault); 
    for (unsigned int i=0; i < B_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    hipHostAlloc((void **) &C_h, C_sz*sizeof(float),hipHostMallocDefault);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    size Of vector: %u x %u\n  ", VecSize,1);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
//    startTime(&timer);
    

    float *A0_d, *B0_d, *C0_d;
    float *A1_d, *B1_d, *C1_d;
    float *A2_d, *B2_d, *C2_d;
    float *A3_d, *B3_d, *C3_d;
    float *A_d, *B_d, *C_d;
    int SegSize=VecSize/4;
    hipMalloc((void**)&A0_d,sizeof(float)*SegSize);
    hipMalloc((void**)&B0_d,sizeof(float)*SegSize);
    hipMalloc((void**)&C0_d,sizeof(float)*SegSize);

    hipMalloc((void**)&A1_d,sizeof(float)*SegSize);
    hipMalloc((void**)&B1_d,sizeof(float)*SegSize);
    hipMalloc((void**)&C1_d,sizeof(float)*SegSize);

    hipMalloc((void**)&A2_d,sizeof(float)*SegSize);
    hipMalloc((void**)&B2_d,sizeof(float)*SegSize);
    hipMalloc((void**)&C2_d,sizeof(float)*SegSize);

    hipMalloc((void**)&A3_d,sizeof(float)*SegSize);
    hipMalloc((void**)&B3_d,sizeof(float)*SegSize);
    hipMalloc((void**)&C3_d,sizeof(float)*SegSize);


    hipMalloc((void**)&A_d,sizeof(float)*4*SegSize);
    hipMalloc((void**)&B_d,sizeof(float)*4*SegSize);
    hipMalloc((void**)&C_d,sizeof(float)*4*SegSize);

    hipDeviceSynchronize();
    // Initialize thread block and kernel grid dimensions ---------------------
    hipStream_t stream0,stream1,stream2,stream3;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    // Timer timer;
     startTime(&timeExe);

     int blocksize=256;
     int num=VecSize/(SegSize*4);
     int i=0;
 
    
    //INSERT CODE HERE
  //  basicVecAdd(A_h,B_h,C_h,VecSize);

    for(int j=0;j<num;j++){

             hipMemcpyAsync(A0_d,A_h+i,SegSize*sizeof(float),hipMemcpyHostToDevice,stream0);
             hipMemcpyAsync(B0_d,B_h+i,SegSize*sizeof(float),hipMemcpyHostToDevice,stream0);
             hipMemcpyAsync(A1_d,A_h+i+SegSize,SegSize*sizeof(float),hipMemcpyHostToDevice,stream1);
             hipMemcpyAsync(B1_d,B_h+i+SegSize,SegSize*sizeof(float),hipMemcpyHostToDevice,stream1);

            VecAdd<<<(SegSize-1)/blocksize+1,blocksize,0,stream0>>>(SegSize,A0_d,B0_d,C0_d);
            VecAdd<<<(SegSize-1)/blocksize+1,blocksize,0,stream1>>>(SegSize,A1_d,B1_d,C1_d);

            hipMemcpyAsync(C_h+i,C0_d,SegSize*sizeof(float),hipMemcpyDeviceToHost,stream0);
            hipMemcpyAsync(C_h+i+SegSize,C1_d,SegSize*sizeof(float),hipMemcpyDeviceToHost,stream1);


             hipMemcpyAsync(A2_d,A_h+i+2*SegSize,SegSize*sizeof(float),hipMemcpyHostToDevice,stream2);
             hipMemcpyAsync(B2_d,B_h+i+2*SegSize,SegSize*sizeof(float),hipMemcpyHostToDevice,stream2);
             hipMemcpyAsync(A3_d,A_h+i+3*SegSize,SegSize*sizeof(float),hipMemcpyHostToDevice,stream3);
             hipMemcpyAsync(B3_d,B_h+i+3*SegSize,SegSize*sizeof(float),hipMemcpyHostToDevice,stream3);


            VecAdd<<<(SegSize-1)/blocksize+1,blocksize,0,stream2>>>(SegSize,A2_d,B2_d,C2_d);
            VecAdd<<<(SegSize-1)/blocksize+1,blocksize,0,stream3>>>(SegSize,A3_d,B3_d,C3_d);


            hipMemcpyAsync(C_h+i+2*SegSize,C2_d,SegSize*sizeof(float),hipMemcpyDeviceToHost,stream2);
            hipMemcpyAsync(C_h+i+3*SegSize,C3_d,SegSize*sizeof(float),hipMemcpyDeviceToHost,stream3);
            i+=4*SegSize;
      }
       if(i<VecSize){
           hipMemcpy(A_d,A_h+i,sizeof(float)*(VecSize-i),hipMemcpyHostToDevice);
           hipMemcpy(B_d,B_h+i,sizeof(float)*(VecSize-i),hipMemcpyHostToDevice);

             VecAdd<<<(VecSize-i-1)/blocksize+1,blocksize>>>(VecSize-i,A_d,B_d,C_d);


             hipMemcpy(C_h+i,C_d,(VecSize-i)*sizeof(float),hipMemcpyDeviceToHost);}

      stopTime(&timeExe);printf("\n Execution time is %f s\n",elapsedTime(timeExe));

  cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
  //  stopTime(&timer); printf("Excution time is: %f s\n", elapsedTime(timer));
    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, VecSize);


    // Free memory ------------------------------------------------------------

    hipHostFree(A_h);
    hipHostFree(B_h);
    hipHostFree(C_h);
   hipFree(A0_d);
    hipFree(B0_d);
    hipFree(C0_d);
    hipFree(A1_d);
    hipFree(B1_d);
    hipFree(C1_d);
   hipFree(A2_d);
    hipFree(B2_d);
    hipFree(C2_d);
    hipFree(A3_d);
    hipFree(B3_d);
    hipFree(C3_d);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    //INSERT CODE HERE
   // hipFree(A0_d);
   // hipFree(B0_d);
   // hipFree(C0_d);
   // hipFree(A1_d);
   // hipFree(B1_d);
   // hipFree(C1_d);   
    return 0;

}
